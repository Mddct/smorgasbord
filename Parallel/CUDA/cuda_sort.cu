
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include<fstream>
#include<sstream>
#include<vector>

using namespace std;

void cdf_intreader(char* fname,int** A, int* n){
	ifstream F(fname);
	stringstream buf;
	buf<< F.rdbuf();
	string S(buf.str());
	
	int lastidx=-1;
	int nextidx=string::npos+1;
	int nread=0;
	string toinsert;
	vector<int> Av;

	int i=0;
	while (nextidx!=string::npos){
		nextidx=S.find(',',lastidx+1);
		if (nextidx!=string::npos){
			toinsert=S.substr(lastidx+1,nextidx-lastidx-1);
			lastidx=nextidx;
		}else{
			toinsert=S.substr(lastidx+1,S.length()-lastidx-1);
		}
		Av.push_back(atoi(toinsert.c_str()));
		i++;
	}

	*n=Av.size();
	*A=new int[Av.size()];
	for (i=0;i<Av.size();i++){
	(*A)[i]=Av[i];
	}
}

//Write seq, which has length n, to fname.
//Also hamelessly copied from Dr. Eicholz...
void cdf_write_seq(char* fname, int* seq, int n){
	ofstream F(fname);
	for (int i=0;i<n-1;i++){
		F<<seq[i]<<",";
	}
	F<<seq[n-1]<<endl;

}

//This section runs on the GPUs
__global__ void kernel(int* arr, int length){
	//What is my ID?
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	//If we're going to access something outside the array, exit
	if(id >= length-1) return;
	
	//Odd/even transpose elements in the list, in parallel (avoiding accessing the same memory)
	for(int j = 0; j < length; j++){
		int temp;
		//If I'm going to perform a swap this round, swap!
		if((j % 2 == 0 && id % 2 == 0) || (j % 2 != 0 && id % 2 != 0)){
			if(arr[id] > arr[id+1]){
				temp = arr[id];
				arr[id] = arr[id+1];
				arr[id+1] = temp;
			}
		}
	}
}

//Main program
int main(int argc, char** argv){
	//Process input files
	if(argc < 3){
		cerr << "Please provide input and output filenames\n";
		abort();
	}
	char* in_file=argv[1];
	char* out_file=argv[2];

	int* A;		//list of integers to be sorted
	int n;		//size of the list of integers
	
	//Bring data in from txt file
	cdf_intreader(in_file,&A,&n);
	
	//Print out initial data if small enough
	if(n < 20){
		cout << "Input list is: ";
		for(int i = 0; i < n-1; i++){
			cout << A[i] << ",";
		}
		cout << A[n-1] << endl;
	}
	
	//How much data is in each thread?
	int bytes = n * sizeof(int);

	//Create pointer to device array
	int *deviceArray;

	//Create the array on the GPU and copy data to it's memory
	hipMalloc((void**)&deviceArray, bytes);
	hipMemcpy(deviceArray, A, bytes, hipMemcpyHostToDevice);
	
	//How many threads and blocks per thread will we have?
	int threads = n/2;
	
	//Launch kernel on the GPU
	kernel<<<n/threads+1,threads>>>(deviceArray,n);
	
	//Gather data back from processors
	hipMemcpy(A, deviceArray, bytes, hipMemcpyDeviceToHost);
	
	//Print output
	if(n < 20){
		cout << "Sorted list is: ";
		for(int i = 0; i < n-1; i++){
			cout << A[i] << ",";
		}
		cout << A[n-1] << endl;
	}
	
	//Write to output file
	cdf_write_seq(out_file,A,n);
	
	//Deallocate the two arrays
	hipFree(deviceArray);
	
	//Exit from the calling program
	return 0;
}