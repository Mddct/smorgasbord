#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "cuPrintf.cu"

__global__ void kernel(void){
	//Print a greeting from the GPU core
	cuPrintf("Hello from GPU processor %d thread %d\n", blockIdx.x, threadIdx.x);
}

int main(int argc, char** argv){
	//How many blocks and how many threads per block will we use?
	int blocks = 2;
	int threadsPerBlock = 2;
	
	//Say Hi from the CPU
	printf("Hello from the CPU\n");
	
	//Initialize printing from GPU cores
	cudaPrintfInit();
	
	//Instruct each GPU core to run its kernel section
	kernel<<<blocks,threadsPerBlock>>>();
	
	//Display the greetings gathered from the GPU cores
	cudaPrintfDisplay();
	
	//End the CUDA printing
	cudaPrintfEnd();
	
	//Exit from the calling program
	return 0;
}