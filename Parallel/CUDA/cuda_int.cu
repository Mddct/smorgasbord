
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

//Note that any functions that want to be called from the kernel must be preceeded with __device__

//Function we are integrating
__device__ float myFunction(float x){
  return pow(x,4);
}

//Trapezoidal rule calculation
__device__ float trapezoidal(float a, float b){
	return (b-a)*((myFunction(a)+myFunction(b))/2);
}

//Composite trap rule calculation
__device__ float composite_trapezoidal(float a, float b, int n){
	float h=(b-a)/(n);
	float total=0;
	int i;
	for (i=0;i<n;i++){
		total=total+trapezoidal(a+i*h,a+(i+1)*h); 
	}
	return total;
}

//This section runs on the GPUs
__global__ void kernel(float* arr, float A, float B, int P, int N){
	//Who am I?
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	//calculate number of intervals, where they start, and where they end, and what interval this processor will use
	float intervalWidth = (B-A)/(P);
	float intervalStart = A+(intervalWidth)*(id);
	float intervalEnd = intervalStart+intervalWidth;

	//calculate the partial sum of this interval
	arr[id] = composite_trapezoidal(intervalStart,intervalEnd,N);
}

int main(int argc, char** argv){
	//Process input from command line
	if (argc<3){
		printf("Please enter a,b,N\n");
		return 1;
	}
	
	float A=atof(argv[1]);
	float B=atof(argv[2]);
	int N=atoi(argv[3]);
	
	printf("Integrating x^4 from %.3f to %.3f with %d points\n", A, B, N);
	
	//How many threads will we use and how much data is in each thread?
	int elements = 512;
	int bytes = elements * sizeof(float);

	//Create pointers to host and device arrays
	float *hostArray = 0;
	float *deviceArray = 0;

	//Create the array on the host and on the GPU
	hostArray = (float*) malloc(bytes);
	hipMalloc((void**)&deviceArray, bytes);

	int blockSize = 128;
	int gridSize = elements / blockSize;

	//Instruct each GPU core to run its kernel section
	kernel<<<gridSize,blockSize>>>(deviceArray, A, B, elements, N);

	//Gather all the partial sums
	hipMemcpy(hostArray, deviceArray, bytes, hipMemcpyDeviceToHost);

	//Reduce the partial sums to a single integral
	float sum = 0;
	for(int i=0; i < elements; ++i){
		sum += hostArray[i];
	}
	
	//Print result
	printf("Integrating x^4 from %.3f to %.3f with %d points is: %.3f\n", A, B, N, sum);

	//Deallocate the two arrays
	free(hostArray);
	hipFree(deviceArray);
	
	//Exit from the calling program
	return 0;
}